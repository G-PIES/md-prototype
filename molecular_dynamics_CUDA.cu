/*
 * Program: 	molecular_dynamics_CUDA.cpp
 * Summary: 	This program is a proof of concept for both a Molecular Dynamic model
 *				as well as a test of various concepts and techniques used with CUDA
 * Programmer:	Sean B. Higgins
 * Start Date:	November 26, 2023
 */

// System libraries to include

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

